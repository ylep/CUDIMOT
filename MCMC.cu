#include "hip/hip_runtime.h"
/* MCMC.cu

   Moises Hernandez-Fernandez - FMRIB Image Analysis Group
   
   Copyright (C) 2005 University of Oxford */

/* CCOPYRIGHT */

// Markoc Chain Monte Carlo method

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MCMC.h"
#include "utils.h"
#include "cudimotoptions.h"
#include "mymodels/mymodel.h"
#include MYMODEL_FUNCTIONS

namespace Cudimot{

#define maxfloat 1e10f
  
  __constant__ float LowerLimits [NPARAMS];
  __constant__ float UpperLimits [NPARAMS];
  
  // SHFL double precision
  /* __device__ inline double __shfl_down(double var, unsigned int srcLane, int width=32) {
    int2 a = *reinterpret_cast<int2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width);
    a.y = __shfl_down(a.y, srcLane, width);
    return *reinterpret_cast<double*>(&a);
    }*/
  
  __device__ inline float log_gpu(float x){return logf(x);}
  __device__ inline double log_gpu(double x){return log(x);}
  __device__ inline float exp_gpu(float x){return expf(x);}
  __device__ inline double exp_gpu(double x){return exp(x);}
  
  // Returns the natural log of the 0th order modified Bessel function of first kind for an argument x
  // Follows the exponential implementation of the Bessel function in Numerical Recipes, Ch. 6
  __device__ inline float logIo(const float x){
    float y,b;
    b=fabsf(x);
    if (b<3.75f){
      float a=x/3.75f;
      a*=a;
      //Bessel function evaluation
      y=1.0f+a*(3.5156229f+a*(3.0899424f+a*(1.2067492f+a*(0.2659732f+a*(0.0360768f+a*0.0045813f)))));
      y=logf(y);
    }else{
      float a=3.75f/b; 
      //Logarithm of Bessel function
      y=b+logf((0.39894228f+a*(0.01328592f+a*(0.00225319f+a*(-0.00157565f+a*(0.00916281f+a*(-0.02057706f+a*(0.02635537f+a*(-0.01647633f+a*0.00392377f))))))))/sqrt(b));
    }
    return y;
  }
  
  __device__ inline double logIo(const double x){
    double y,b;
    b=fabs(x);
    if (b<3.75){
      double a=x/3.75;
      a*=a;
      //Bessel function evaluation
      y=1.0+a*(3.5156229+a*(3.0899424+a*(1.2067492+a*(0.2659732+a*(0.0360768+a*0.0045813)))));
      y=log(y);
    }
    else{
      double a=3.75/b; 
      //Logarithm of Bessel function
      y=b+log((0.39894228+a*(0.01328592+a*(0.00225319+a*(-0.00157565+a*(0.00916281+a*(-0.02057706+a*(0.02635537+a*(-0.01647633+a*0.00392377))))))))/sqrt(b));
    }
    return y;
  }


  template <typename T>
  __device__ inline void Propose(int par, T* params, T* old, T* propSD, hiprandState* localrandState){
    *old=params[par];
    params[par] = params[par] + hiprand_normal(localrandState)*propSD[par];
  }

  template <typename T>
  __device__ inline int Check_limits(int idpar, T* params){
    //if(params[idpar]<LowerLimits[idpar]) return 0;
    //if(params[idpar]>UpperLimits[idpar]) return 0;
    return 1;
  }
  
  
  template <typename T>
  __device__ inline void Check_prior(){
     // check lower limit

     // check upper limit

     // Gaussian (mean,sd)
     // if (param<=0.0f || m_f[fibre]>=1.0f)
    
     // Gamma (alpha,beta)
     
     //(1.0-alpha)* log(param) + beta*param;

     // ARD - Automatic Relevance Determination

     // Beta (alpha,beta)
    
    // sin()

  }
  
  
  template <typename T, bool RICIAN_NOISE>
  __device__ void  Compute_Likelihood(int idSubVOX,
				      int nmeas,
				      int CFP_Tsize,
				      T* measurements,
				      T* parameters,
				      T* tau,
				      T* fixed_params,
				      T* CFP,
				      T* likelihood)
  {
    int idMeasurement=idSubVOX;
    int nmeas2compute = nmeas/THREADS_VOXEL;
    if (idSubVOX<(nmeas%THREADS_VOXEL)) nmeas2compute++;
    
    T accumulated_error=0;
    for(int dir=0;dir<nmeas2compute;dir++){
      T* myCFP = &CFP[idMeasurement*CFP_Tsize];
      T pred_error=Predicted_Signal(NPARAMS,parameters,fixed_params,myCFP);
      if(RICIAN_NOISE){
	T meas = measurements[idMeasurement];
	pred_error=log_gpu(meas)+(-0.5*(*tau)*(meas*meas+pred_error*pred_error)+logIo((*tau)*pred_error*meas));
	accumulated_error+=pred_error;
      }else{
	pred_error=pred_error-measurements[idMeasurement];
	accumulated_error+=pred_error*pred_error;
      }
      idMeasurement+=THREADS_VOXEL;
    }
   
    for (int offset=THREADS_VOXEL/2; offset>0; offset/=2){
      accumulated_error+= __shfl_down(accumulated_error,offset);
    }
    
    if(idSubVOX==0){
      if(RICIAN_NOISE){
	*likelihood = -nmeas*log_gpu(*tau)-accumulated_error;
      }else{
	*likelihood = (nmeas/2.0f)*log_gpu(accumulated_error/2.0f);
      }
    }
  }
  
  template <typename T>
  __device__ inline int Compute_test_energy(T* new_energy, T* old_energy, T* prior, T* likelihood, hiprandState* localrandState){
    (*old_energy) = (*new_energy);
    (*new_energy) = (*prior)+ (*likelihood);
    
    T tmp=exp_gpu((*old_energy)-(*new_energy));
    
    return (tmp>hiprand_uniform(localrandState));
  }
  

  template <typename T, bool RICIAN_NOISE, bool RECORDING>
  __global__ void mcmc_kernel(
			      hiprandState* randstate, // to generate random numbers
			      int nvox, // num voxels
			      int nmeas, // num measurements
			      int CFP_Tsize, //size*M-measurements
			      int niters,
			      int nsamples, // num samples per parameter
			      int sampleevery, // record a sample every x iterations
			      int updateproposalevery, // update SD proposals every x iters
			      T* meas, // measurements
			      T* parameters, // model parameters 
			      T* CFP, // common fixed model parameters
			      T* samples) // to record parameters samples
  {
    // 1 block of threads process several voxels
    // Each warp processes 1 voxel
    int idVOX= (blockIdx.x*VOXELS_BLOCK)+int(threadIdx.x/THREADS_VOXEL);
    int idVOX_inBlock =  threadIdx.x/THREADS_VOXEL;
    int idSubVOX= threadIdx.x%THREADS_VOXEL;
    bool leader = (idSubVOX==0);  // Some steps are performed by only one thread of the warp
    
    ////////// DYNAMIC SHARED MEMORY ///////////
    extern __shared__ double shared[];		     			// Size: 
    hiprandState* localrandState = (hiprandState*)shared;		

    T* S_CFP = (T*) &localrandState[VOXELS_BLOCK]; 	       	        // nmeas*CFP_Tsize
    T* params = (T*) &S_CFP[nmeas*CFP_Tsize]; 				// NPARAMS*VOXELS_BLOCK 
    T* priors = (T*) &params[NPARAMS*VOXELS_BLOCK]; 			// NPARAMS*VOXELS_BLOCK
    T* propSD = (T*) &priors[NPARAMS*VOXELS_BLOCK]; 			// NPARAMS*VOXELS_BLOCK
    
    T* likelihood = (T*) &propSD[NPARAMS*VOXELS_BLOCK]; 		// VOXELS_BLOCK 
    T* Tprior = (T*) &likelihood[VOXELS_BLOCK]; 			// VOXELS_BLOCK 
    T* energy = (T*) &Tprior[VOXELS_BLOCK];				// VOXELS_BLOCK 
    T* tau = (T*) &energy[VOXELS_BLOCK];			       	// VOXELS_BLOCK 

    T* old_param = (T*) &tau[VOXELS_BLOCK];		 	      	// VOXELS_BLOCK 
    T* old_energy =  (T*) &old_param[VOXELS_BLOCK];		       	// VOXELS_BLOCK 

    int* naccepted = (int*) &old_energy[VOXELS_BLOCK];			//NPARAMS*VOXELS_BLOCK
    int* nrejected = (int*) &naccepted[NPARAMS*VOXELS_BLOCK];		//NPARAMS*VOXELS_BLOCK
    ////////////////////////////////////////////
    
    /// Copy common fixed model parameters to Shared Memory ///
    if(threadIdx.x==0){ // only one thread of the whole block. Common to all voxels
      for(int i=0;i<nmeas*CFP_Tsize;i++){
	S_CFP[i]=CFP[i];
      }
    }
    ///////////////////////////////////////////////////////////
    
    ///////// each voxel/warp of the block points to its data///////////
    meas = &meas[idVOX*nmeas]; // Global memory
    if(RECORDING){
      samples = &samples[idVOX*NPARAMS*nsamples]; //Global memory
    }
    localrandState = (hiprandState*)&localrandState[idVOX_inBlock];
    params = &params[idVOX_inBlock*NPARAMS];
    priors = &priors[idVOX_inBlock*NPARAMS];
    propSD = &propSD[idVOX_inBlock*NPARAMS];
    likelihood = &likelihood[idVOX_inBlock];
    Tprior = &Tprior[idVOX_inBlock];
    energy = &energy[idVOX_inBlock];
    tau = &tau[idVOX_inBlock];
    old_param = &old_param[idVOX_inBlock];
    old_energy = &old_energy[idVOX_inBlock];
    naccepted = &naccepted[idVOX_inBlock*NPARAMS];
    nrejected = &nrejected[idVOX_inBlock*NPARAMS];
    
    /// Ititialise shared values of each voxel: only the leader///
    if(leader){ 
      *localrandState = randstate[idVOX];
      for(int par=0;par<NPARAMS;par++){
      	params[par]=parameters[idVOX*NPARAMS+par];
	naccepted[par]=0;
	nrejected[par]=0;
	priors[par]=0;
	propSD[par]=params[par]/10.0; // add by the user
      }
      *tau=0;
      *Tprior=0;
    }
    __syncthreads();
    ///////////////////////////////////////////
    
    T* fixed_params=new T[1]; // TODO: add feature
    
    Compute_Likelihood<T,RICIAN_NOISE>(idSubVOX,nmeas,CFP_Tsize,meas,params,tau,fixed_params,S_CFP,likelihood);
    if(leader){
      *energy=(*Tprior)+(*likelihood);
    }
    
    for (int iter=0; iter<niters; iter++){
      for (int par=0; par<NPARAMS; par++){
	int criteria=0;
	if(leader){
	  Propose(par,params,old_param,propSD,localrandState);
	  criteria=Check_limits(par,params);
	}
	criteria = __shfl(criteria,0);
	
	if(criteria){
	  Compute_Likelihood<T,RICIAN_NOISE>(idSubVOX,nmeas,CFP_Tsize,meas,params,tau,fixed_params,S_CFP,likelihood);
	  if(leader){
	    criteria=Compute_test_energy(energy,old_energy,Tprior,likelihood,localrandState);
	    if(criteria){
	      naccepted[par]++;
	    }else{
	      nrejected[par]++;
	      params[par]=(*old_param);
	      *energy=*old_energy;
	    }
	  }
	  criteria = __shfl(criteria,0); // better than __syncthreads(); ?
	}else{
	  nrejected[par]++;
	  params[par]=(*old_param);
	}
      }
      
      if(RECORDING){
	if((!(iter%sampleevery))&&(leader)){
	  int nsamp=iter/sampleevery;
	  for (int par=0; par<NPARAMS; par++){
	    samples[par*nsamples+nsamp]=params[par];
	  }
	}
      }

      if(!RECORDING){  // deactivated when not recording ?
	if((iter>0)&&(!(iter%updateproposalevery))&&(leader)){
	  for (int par=0; par<NPARAMS; par++){
	    propSD[par]=sqrt((naccepted[par]+1.0)/(nrejected[par]+1.0)); // not too big ?
	    propSD[par]=min(propSD[par],maxfloat);
	  }
	}
      }
    }
    if(leader){
      randstate[idVOX]=*localrandState; // save state, otherwise random numbers will be repeated (start at the same point)
    }
  }
  
  __global__ void setup_randoms_kernel(hiprandState* randstate, double seed){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init(seed,id,0,&randstate[id]);
  }
  
  template <typename T>
  MCMC<T>::MCMC(int nvoxFitpart){
    cudimotOptions& opts = cudimotOptions::getInstance();
    nvoxFit_part=nvoxFitpart;
    nburnin=opts.nburn.value();
    njumps=opts.njumps.value();
    nsamples=opts.njumps.value()/opts.sampleevery.value();
    sampleevery=opts.sampleevery.value();
    updateproposalevery=opts.sampleevery.value();
    RicianNoise=opts.rician.value();
    
    // Initialise Randoms
    int blocks_Rand = nvoxFit_part/256;
    if(nvoxFit_part%256) blocks_Rand++;
    hipMalloc((void**)&randStates, blocks_Rand*256*sizeof(hiprandState));
    dim3 Dim_Grid_Rand(blocks_Rand,1);
    dim3 Dim_Block_Rand(256,1);
    srand(opts.seed.value());  //randoms seed
    setup_randoms_kernel<<<Dim_Grid_Rand,Dim_Block_Rand>>>(randStates,rand());
    sync_check("Setup_Randoms_kernel");
  }
  
  template <typename T>
  void MCMC<T>::run(
		    int nvox, int nmeas,
		    int CFP_size,
		    T* meas,
		    T* params,
		    T* CFP,
		    T* samples) 
  {
    long int amount_shared_mem = 0;
    amount_shared_mem += VOXELS_BLOCK*sizeof(hiprandState); // hiprandState
    amount_shared_mem += (nmeas*CFP_size)*sizeof(T); // CFP
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // Parameters
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // Priors
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // PropSD
    amount_shared_mem += (4*VOXELS_BLOCK)*sizeof(T); // Likelihod,TPrior,Energy, Tau
    amount_shared_mem += (2*VOXELS_BLOCK)*sizeof(T); // old_param, old_energy
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(int); // naccepted
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(int); // nrejected

    cout << "Shared Memory used in MCMC kernel: " << amount_shared_mem << endl;
    
    int threads_block = VOXELS_BLOCK * THREADS_VOXEL;
    int nblocks=(nvox/VOXELS_BLOCK);
    if(nvox%VOXELS_BLOCK) nblocks++;
    
    // Burn-In
    if(RicianNoise){
      mcmc_kernel<T,true,false><<<nblocks,threads_block,amount_shared_mem>>>(randStates,nvox,nmeas,CFP_size,nburnin,nsamples,sampleevery,updateproposalevery,meas,params,CFP,samples);
    }else{
      mcmc_kernel<T,false,false><<<nblocks,threads_block,amount_shared_mem>>>(randStates,nvox,nmeas,CFP_size,nburnin,nsamples,sampleevery,updateproposalevery,meas,params,CFP,samples);
    }
    sync_check("MCMC Kernel: burnin step");
    
    // Recordig
    if(RicianNoise){
      mcmc_kernel<T,true,true><<<nblocks,threads_block,amount_shared_mem>>>(randStates,nvox,nmeas,CFP_size,njumps,nsamples,sampleevery,updateproposalevery,meas,params,CFP,samples);
    }else{
      mcmc_kernel<T,false,true><<<nblocks,threads_block,amount_shared_mem>>>(randStates,nvox,nmeas,CFP_size,njumps,nsamples,sampleevery,updateproposalevery,meas,params,CFP,samples);
    }
    sync_check("MCMC Kernel: recording step"); 
    
    printf("--------------------- MCMC completed ------------\n");
  }

  // Explicit Instantiations of the template
  template class MCMC<float>;
  template class MCMC<double>;
}
