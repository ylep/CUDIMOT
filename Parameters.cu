/* Parameters.cu
   
   Moises Hernandez-Fernandez - FMRIB Image Analysis Group
   
   Copyright (C) 2005 University of Oxford */

/* CCOPYRIGHT */

#include "Parameters.h"

namespace Cudimot{
  
  template <typename T>
  Parameters<T>::Parameters(Model<T> model,dMRI_Data<T> dMRI_data):
    nparams(model.nparams),
    nFP(model.nFP), nCFP(model.nCFP),CFP_Tsize(model.CFP_Tsize),
    nvox(dMRI_data.nvox), nmeas(dMRI_data.nmeas),nparts(dMRI_data.nparts),
    size_part(dMRI_data.size_part),size_last_part(dMRI_data.size_last_part),
    nvoxFit_part(dMRI_data.nvoxFit_part)
  {
    
    Log& logger = LogSingleton::getInstance();
    cudimotOptions& opts = cudimotOptions::getInstance();
    
    //////////////////////////////////////////////////////
    /// Initialise parameters
    /// The user can provide nifti files for some parameters
    //////////////////////////////////////////////////////
    params_host=new T[nvox*nparams];
    
    if (opts.init_params.set()){
      // If volumes provided for initialize parameters
      for(int idParam=0;idParam<nparams;idParam++){
	// Read binary file with values for this part, logfile
	string name_file;
	name_file.append(opts.partsdir.value());
	name_file.append("/part_");
	name_file.append(num2str(opts.idPart.value()));
	name_file.append("/Param_");
	name_file.append(num2str(idParam));
	
	ifstream in;
	long nbytes;
	int nvox_file,nmeas_file;
	in.open(name_file.data(), ios::in | ios::binary);
	in.read((char*)&nvox_file, 4);
	in.read((char*)&nmeas_file, 4);
	in.read((char*)&nbytes, sizeof(long));
	
	if(nvox!=nvox_file || nmeas_file!=1){
	  cerr << "CUDIMOT Error: The amount of data in the input file " <<  name_file << " for initializing the parameters is not correct" << endl;
	  exit(-1);
	}
	
	Matrix Parameters_init;
	Parameters_init.ReSize(1,nvox);
	in.read((char*)&Parameters_init(1,1),nbytes);
	in.close();
	
	for(int i=0;i<nvox;i++){
	  params_host[i*nparams+idParam]=Parameters_init(1,i+1);
	}
      }
      
    }else{
      // If NOT volumes provided for initialize parameters, try default values
      for(int idParam=0;idParam<nparams;idParam++){
	for(int i=0;i<nvox;i++){
	  if(model.initProvided()){
	    params_host[i*nparams+idParam]=model.getParam_init(idParam);
	  }else{
	    params_host[i*nparams+idParam]=0;
	  }
	}
      }
    }
    
    //////////////////////////////////////////////////////
    /// Read Common Fixed Parameters (kxM, M:measurements)
    /// Provided by the user
    /// If not provided use the defaut values (given by the model-designer)
    //////////////////////////////////////////////////////
    CFP_host= new T[nmeas*CFP_Tsize];
    
    if (opts.CFP.set()){
      int nCFP_set = 0; //to check that all the values are set
      int cumulative_size=0;
      string filename(opts.CFP.value());
      ifstream file(filename.data());
      if (file.is_open()){
	string line;
	while(getline(file,line)){
       	  if (!line.empty()){ // if is empty, what should I do?
	    // Read file with values
	    Matrix values;
	    values=read_ascii_matrix(line);
	    int param_size = model.CFP_sizes[nCFP_set];
	    if(values.Nrows()!=param_size){
	      cerr << "CUDIMOT Error: Common Fixed Parameter number " << nCFP_set << " and file " << line << " do not match the dimensions specified for this model" << endl;
	      exit(-1);
	    }
	    if(values.Ncols()!=nmeas){
	      cerr << "CUDIMOT Error:  Common Fixed Parameter number " << nCFP_set << " and file " << line << " do not match the number of measurements: " << nmeas << endl;
	      exit(-1);
	    }
	    
	    for(int i=0;i<param_size;i++){
	      for (int j=0;j<nmeas;j++){
		CFP_host[cumulative_size+i+j*CFP_Tsize]=values(i+1,j+1);
	      }
	    }
	    nCFP_set++;
	    cumulative_size+=param_size;
	    
	  }
	} //end lines
	
	if(nCFP_set!=nCFP){
	  cerr << "CUDIMOT Error: The number of common fixed parameter provided in file: " << filename.data() << " is not correct for this model. The number of common fixed parameter must be " << nCFP << endl;
	  exit(-1);
	}
      }else{
	cerr << "CUDIMOT Error: Unable to open Common Fixed Parameters file: " << filename.data() << endl; 
	exit(-1);
      }
    
    }else{
      // No CFP provided ?
      
    }
    //////////////////////////////////////////////////////
    
        
    /// Allocate GPU memory
    hipMalloc((void**)&params_gpu,nvoxFit_part*nparams*sizeof(T));
    hipMalloc((void**)&CFP_gpu,CFP_Tsize*nmeas*sizeof(T));
    sync_check("Allocating Model Parameters on GPU\n");
    
    // Copy Parameters from host to GPU
    hipMemcpy(CFP_gpu,CFP_host,CFP_Tsize*nmeas*sizeof(T),hipMemcpyHostToDevice);
    sync_check("Copying Common-Fixed Model Parameters to GPU\n");
    
    /// If MCMC: allocate memory in host and GPU for samples;
    if(opts.runMCMC.value()){
      nsamples=(opts.njumps.value()/opts.sampleevery.value());   
    }else{
      nsamples=1;
    }
    samples_host = new T[nsamples*nparams*nvox];
    hipMalloc((void**)&samples_gpu,nvoxFit_part*nparams*nsamples*sizeof(T));
  }
  
  template <typename T>
  Parameters<T>::~Parameters(){}
  
  template <typename T>
  T* Parameters<T>::getParametersPart(int part){
    if(part>=nparts){
      cerr << "CUDIMOT Error: Trying to get an incorrect part of the Parameters: " << part << ". There are only " << nparts << " parts and index starts at 0." << endl;
      exit(-1);
    }
    
    int initial_pos=part*size_part*nparams;
    // Copy from host to GPU
    hipMemcpy(params_gpu,&params_host[initial_pos],nvoxFit_part*nparams*sizeof(T),hipMemcpyHostToDevice);
    sync_check("Copying Model Parameters to GPU\n");

    return params_gpu;
  }
 
  template <typename T>
  T* Parameters<T>::getSamples(){
    return samples_gpu;
  }

  template <typename T>
  int Parameters<T>::getTsize_CFP(){
    return CFP_Tsize;
  }
  
  template <typename T>
  T* Parameters<T>::getCFP(){
    return CFP_gpu;
  }
  
  template <typename T>
  void Parameters<T>::copyParamsPartGPU2Host(int part){

    if(part>=nparts){
      cerr << "CUDIMOT Error: Trying to store an incorrect part of Parameters: " << part << ". There are only " << nparts << " parts and index starts at 0." << endl;
      exit(-1);
    }
    
    int size=size_part; // this ignores the extra voxels added
    if(part==(nparts-1)){
      size=size_last_part; // this ignores the extra voxels added
    }
    int initial_pos=part*size_part*nparams;
    
    hipMemcpy(&params_host[initial_pos],params_gpu,size*nparams*sizeof(T),hipMemcpyDeviceToHost);
     sync_check("Copying Model Parameters from GPU\n");
     
     samples_host=params_host; 
  }
  
  template <typename T>
  void Parameters<T>::copyParams2Samples(){
    samples_host=params_host;
  }
  
  
  template <typename T>
  void Parameters<T>::copySamplesPartGPU2Host(int part){
    if(part>=nparts){
      cerr << "CUDIMOT Error: Trying to store an incorrect part of Samples: " << part << ". There are only " << nparts << " parts and index starts at 0." << endl;
      exit(-1);
    }
    
    int size=size_part; // this ignores the extra voxels added
    if(part==(nparts-1)){
      size=size_last_part; // this ignores the extra voxels added
    }
    int initial_pos=part*size_part*nparams*nsamples;
    
    hipMemcpy(&samples_host[initial_pos],samples_gpu,size*nparams*nsamples*sizeof(T),hipMemcpyDeviceToHost);
    sync_check("Copying Samples from GPU\n");
  }
  
  template <typename T>
  void Parameters<T>::writeSamples(){
    Log& logger = LogSingleton::getInstance();
    cudimotOptions& opts = cudimotOptions::getInstance();

    // Create a Matrix [nsamples X nvoxels] for each parameter
    vector<Matrix> samples;
    samples.resize(nparams);
    
    for(int par=0;par<nparams;par++){
      samples[par].ReSize(nsamples,nvox);
      samples[par]=0;
    }
    // Copy samples to each Matrix
    for(int vox=0;vox<nvox;vox++){  
      for(int par=0;par<nparams;par++){
	for(int sam=0;sam<nsamples;sam++){
	  samples[par](sam+1,vox+1)=samples_host[vox*nparams*nsamples+par*nsamples+sam];
	}
      }
    }
    
    // Write to file
    for(int par=0;par<nparams;par++){
      string file_name;
      file_name.append(opts.partsdir.value());
      file_name.append("/part_");
      file_name.append(num2str(opts.idPart.value()));
      file_name.append("/Param_"+num2str(par)+"_samples");
      ofstream out;
      out.open(file_name.data(), ios::out | ios::binary);
      out.write((char*)&nvox,4); // number of voxels
      out.write((char*)&nsamples,4); // number of measurements
      long size=nvox*nsamples*sizeof(Real); //need Real here (NEWMAT Object!)
      out.write((char*)&size,sizeof(long)); // number of bytes
      out.write((char*)&samples[par](1,1),size);
      out.close();
    }
  }
  
  // Explicit Instantiations of the template
  template class Parameters<float>;
  template class Parameters<double>;
}
