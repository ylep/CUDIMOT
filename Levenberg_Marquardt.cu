#include "hip/hip_runtime.h"
/* Levenberg_Marquardt.cu

   Moises Hernandez-Fernandez - FMRIB Image Analysis Group
   
   Copyright (C) 2005 University of Oxford */

/* CCOPYRIGHT */

#include "Levenberg_Marquardt.h"
#include "utils.h"
#include "cudimotoptions.h"
#include "mymodels/mymodel.h"
#include MYMODEL_FUNCTIONS

namespace Cudimot{

#define CFTOL 1.0e-8
#define LTOL 1.0e20
#define EPS_gpu 2.0e-16 //Losely based on NRinC 20.1
  
  __device__ inline bool zero_cf_diff_conv(double* cfo,double* cfn){
    return(2.0*fabs(*cfo-*cfn) <= CFTOL*(fabs(*cfo)+fabs(*cfn)+EPS_gpu));
  }
  
  template <typename T>
  __device__ void Cost_Function(int idSubVOX,
				int nmeas,
				int CFP_Tsize,
				T* measurements,
				T* parameters,
				T* fixed_params,
				T* CFP,
				double* result)
  {
    int idMeasurement=idSubVOX;
    T accumulated_error=0;

    int nmeas2compute = nmeas/THREADS_VOXEL;
    if (idSubVOX<(nmeas%THREADS_VOXEL)) nmeas2compute++;

    for(int iter=0;iter<nmeas2compute;iter++){
      T* myCFP = &CFP[idMeasurement*CFP_Tsize];
      T pred_error=Predicted_Signal(NPARAMS,parameters,fixed_params,myCFP);
      
      pred_error=pred_error-measurements[idMeasurement];
      accumulated_error+=pred_error*pred_error;
      idMeasurement+=THREADS_VOXEL;
    }
    
    for (int offset=THREADS_VOXEL/2; offset>0; offset/=2){
      accumulated_error+= __shfl_down(accumulated_error,offset);
    }
    if(idSubVOX==0){
      *result=accumulated_error;
    }
  }
  
  template <typename T>
  __device__ void Calculate_Gradient(int idSubVOX,
				     int nmeas,
				     int CFP_Tsize,
				     T* measurements,
				     T* parameters,
				     T* fixed_params,
				     T* CFP,
				     T* Gradient)
  {
    int idMeasurement=idSubVOX;
    T myderivatives[NPARAMS];
    
    int max_iters = nmeas/THREADS_VOXEL;
    if(nmeas%THREADS_VOXEL) max_iters++;
    
    if(idSubVOX==0){
      // #pragma unroll NPARAMS ... nvcc does not like this
      // warning: extra characters in the unroll pragma (expected a single positive integer), ignoring pragma for this loop
#pragma unroll
      for(int p=0;p<NPARAMS;p++){
	Gradient[p]=0;
      }
    }
    
    for(int iter=0;iter<max_iters;iter++){
      for(int p=0;p<NPARAMS;p++){
     	myderivatives[p]=0; // Maybe idMeasurement > nmeas
      }
      T* myCFP = &CFP[idMeasurement*CFP_Tsize];
      T pred_error=0;

      if(idMeasurement<nmeas){
	pred_error=Predicted_Signal(NPARAMS,parameters,fixed_params,myCFP);
	pred_error=pred_error-measurements[idMeasurement];
	
	Partial_Derivatives(NPARAMS,parameters,fixed_params,myCFP,myderivatives);
      }
      
      // #pragma unroll NPARAMS ... nvcc does not like this
#pragma unroll 
      for(int p=0;p<NPARAMS;p++){
	myderivatives[p]=2.0*pred_error*myderivatives[p];
	for (int offset=THREADS_VOXEL/2; offset>0; offset/=2){
	  myderivatives[p]+= __shfl_down(myderivatives[p],offset);
	}
      }
      if(idSubVOX==0){
	// #pragma unroll NPARAMS ... nvcc does not like this
#pragma unroll
	for(int p=0;p<NPARAMS;p++){
	  Gradient[p]+=myderivatives[p];
	}
      }
      idMeasurement+=THREADS_VOXEL;
    }  
  }

  template <typename T>
  __device__ void Calculate_Hessian(int idSubVOX,
				    int nmeas,
				    int CFP_Tsize,
				    T* measurements,
				    T* parameters,
				    T* fixed_params,
				    T* CFP,
				    T* Hessian)
  {
    int idMeasurement=idSubVOX;
    T myderivatives[NPARAMS];
    
    int max_dir = nmeas/THREADS_VOXEL;
    if(nmeas%THREADS_VOXEL) max_dir++;
    
    if(idSubVOX==0){
      for(int p=0;p<NPARAMS;p++){
	for(int p2=0;p2<NPARAMS;p2++){
	  Hessian[p*NPARAMS+p2]=0;
	}
      }
    }

    for(int iter=0;iter<max_dir;iter++){

      T* myCFP = &CFP[idMeasurement*CFP_Tsize];

      for(int p=0;p<NPARAMS;p++){
	myderivatives[p]=0;
      }
      T pred_error=0;

      if(idMeasurement<nmeas){
	pred_error=Predicted_Signal(NPARAMS,parameters,fixed_params,myCFP);
	pred_error=pred_error-measurements[idMeasurement];
	Partial_Derivatives(NPARAMS,parameters,fixed_params,myCFP, myderivatives);
      }
      
// #pragma unroll NPARAMS ... nvcc does not like this
#pragma unroll
      for(int p=0;p<NPARAMS;p++){
	for(int p2=0;p2<NPARAMS;p2++){
	  T element = 2.0 * myderivatives[p] * myderivatives[p2];
	  for (int offset=THREADS_VOXEL/2; offset>0; offset/=2){
	    element+= __shfl_down(element,offset);
	  }
	  if(idSubVOX==0){
	    Hessian[p*NPARAMS+p2]+=element;
	  }
	}
      }
      idMeasurement+=THREADS_VOXEL;
    }  
  }
  
  template <typename T>
  __device__ void LUsolver(int idSubVOX,
			   T* Hessian,
			   T* Gradient,
			   T* Solution){
    
    // If NPARAMS > 32 the current version of the method will fail !!
    // Need to generalise
    
    T col_elems[NPARAMS];
    T pivot;
    
    // Initialise Matrix. Each thread contains a column of the Hessian and one thread the Gradient column:   Matrix = [Hessian | Gradient]
    if (idSubVOX<NPARAMS){
      for(int p=0;p<NPARAMS;p++){
	col_elems[p] = Hessian[p*NPARAMS+idSubVOX];
      }
    }else if(idSubVOX==NPARAMS){
      for(int p=0;p<NPARAMS;p++){
	col_elems[p] = Gradient[p];
      }
    }else{
      for(int p=0;p<NPARAMS;p++){
	col_elems[p] = 0;
      }
    }
    
    // Solve in two steps: 
    // Forward step: Zero's under diagonal
    // Backward step: Zero's above diagonal

    // Forward step
    for (int col=0; col<NPARAMS; col++){
      // Divide row by diagonal element (1 in the diagonal)
      // Cannot have negative numbers in the diagonal -r/-r = +1
      pivot = col_elems[col];
      pivot = __shfl(pivot,col);
      col_elems[col] = col_elems[col]/pivot; 
      
      // Eliminate all terms under diagonal element (1)
      // Pivot is the element to make zero, Pivot-Pivot*1 = 0
      // This_row = This_row - Pivot * row_of_diagonal_element
      for (int row=col+1; row<NPARAMS; row++){
	pivot  = col_elems[row];
	pivot  = __shfl(pivot,col);
	col_elems[row] -= pivot*col_elems[col];
      }
    }

    // Backward step
    for (int col=NPARAMS-1; col>0; col--) {
      // Eliminate all terms above diagonal element
      for (int row=0; row<col; row++) {
	pivot  = col_elems[row];
	pivot  = __shfl(pivot,col);
	col_elems[row] -= pivot*col_elems[col];
      }
    }

    if(idSubVOX==NPARAMS){
      for(int p=0;p<NPARAMS;p++){
	Solution[p] = col_elems[p];
      }
    }
    
  }
  
  template <typename T, bool MARQUARDT>
  __global__ void levenberg_kernel(
				   int nvox, // nvoxels
				   int nmeas, // nmeasurements
				   int CFP_Tsize, //size*M-measurements
				   T* meas, // measurements
				   T* parameters, // model parameters 
				   T* CFP, // common fixed model parameters
				   int nmax_iters)
{
  // 1 block of threads process several voxels
  // Each warp processes 1 voxel
  int idVOX= (blockIdx.x*VOXELS_BLOCK)+int(threadIdx.x/THREADS_VOXEL);
  int idVOX_inBlock =  threadIdx.x/THREADS_VOXEL;
  int idSubVOX= threadIdx.x%THREADS_VOXEL;
  bool leader = (idSubVOX==0);  // Some steps are performed by only one thread of the warp
    
  ////////// DYNAMIC SHARED MEMORY ///////////
  extern __shared__ double shared[];				//Size:
  double* pcf = (double*) shared;				//VOXELS_BLOCK 
  double* ncf = (double*) &pcf[VOXELS_BLOCK];			//VOXELS_BLOCK
  double* lambda = (double*) &ncf[VOXELS_BLOCK];		//VOXELS_BLOCK
  double* olambda = (double*) &lambda[VOXELS_BLOCK];		//VOXELS_BLOCK
  
  T* S_CFP = (T*) &olambda[VOXELS_BLOCK];			//nmeas*CMP_Tsize
  T* params = (T*) &S_CFP[nmeas*CFP_Tsize]; 			//NPARAMS*VOXELS_BLOCK
  T* Gradient = (T*) &params[NPARAMS*VOXELS_BLOCK];		//NPARAMS*VOXELS_BLOCK
  T* Hessian = (T*) &Gradient[NPARAMS*VOXELS_BLOCK];		//NPARAMS*NPARAMS*VOXELS_BLOCK
  T* step = (T*) &Hessian[NPARAMS*NPARAMS*VOXELS_BLOCK];	//NPARAMS*VOXELS_BLOCK
  
  int* success = (int*) &step[NPARAMS*VOXELS_BLOCK];		//VOXELS_BLOCK
  int* end = (int*) &success[VOXELS_BLOCK];			//VOXELS_BLOCK
  ////////////////////////////////////////////
  
  /// Copy common fixed model parameters to Shared Memory ///
  if(threadIdx.x==0){ // only one thread of the whole block. Common to all voxels
    for(int i=0;i<nmeas*CFP_Tsize;i++){
      S_CFP[i]=CFP[i];
    }
  }
  ///////////////////////////////////////////////////////////

  ///////// each voxel/warp of the block points to its data///////////
  meas = &meas[idVOX*nmeas]; //Global memory
  pcf = &pcf[idVOX_inBlock];
  ncf = &ncf[idVOX_inBlock];
  lambda = &lambda[idVOX_inBlock];
  olambda = &olambda[idVOX_inBlock];
  params = &params[idVOX_inBlock*NPARAMS];
  Gradient = &Gradient[idVOX_inBlock*NPARAMS];
  Hessian = &Hessian[idVOX_inBlock*NPARAMS*NPARAMS];
  step = &step[idVOX_inBlock*NPARAMS];
  success = &success[idVOX_inBlock];
  end = &end[idVOX_inBlock];

  int iter=0;
    
  /// Ititialise shared values of each voxel: only the leader///
  if(leader){ 
    *end=false;
    *success=true;
    *lambda=0.1;
    *olambda= 0.0;    
    *ncf=0.0;
    for(int i=0;i<NPARAMS;i++){
      params[i]=parameters[idVOX*NPARAMS+i];
    }
  }
  __syncthreads();
  ///////////////////////////////////////////

  T* fixed_params=new T[1]; // TODO: add feature
  Cost_Function(idSubVOX,nmeas,CFP_Tsize,meas,params,fixed_params,S_CFP,pcf);
  
  while (!( (*success) && iter++>=nmax_iters)){ 
    //if success we don't increase niter (first condition is true)
    //function cost has been decreased, we have advanced.
    if(*success){
      Calculate_Gradient(idSubVOX,nmeas,CFP_Tsize,meas,params,fixed_params,S_CFP,Gradient);
      Calculate_Hessian(idSubVOX,nmeas,CFP_Tsize,meas,params,fixed_params,S_CFP,Hessian);
    }
    
    if(leader){
      for (int i=0; i<NPARAMS; i++){
	if(MARQUARDT)
	  Hessian[(i*NPARAMS)+i]=((1+(*lambda))/(1+(*olambda)))*Hessian[i*NPARAMS+i];	//Levenberg-Marquardt
	else
	  Hessian[(i*NPARAMS)+i]+=(*lambda)-(*olambda);	//Levenberg
      }
    }
    
    LUsolver(idSubVOX,Hessian,Gradient,step);

    if(leader){
      for(int i=0;i<NPARAMS;i++){
	step[i]=params[i]-step[i];
      }
    }
    __syncthreads();
    
    Cost_Function(idSubVOX,nmeas,CFP_Tsize,meas,step,fixed_params,S_CFP,ncf);

    if(leader){
      if ( *success = ((*ncf) < (*pcf))){ 
	*olambda = 0.0;
	for(int i=0;i<NPARAMS;i++){
	  params[i]=step[i];
	}
	*lambda=(*lambda)/10.0;
	
	if (zero_cf_diff_conv(pcf,ncf)){
	  *end=true;
	}
	*pcf=*ncf;
      }else{
	*olambda=*lambda;
	*lambda=(*lambda)*10.0;
	if(*lambda > LTOL){ 
	 *end=true;
	}
      }
    }	
    __syncthreads();
    if(*end) break;		
  }
  if(leader){
    for(int i=0;i<NPARAMS;i++){
      parameters[idVOX*NPARAMS+i]=params[i];
    }
  }
  __syncthreads();
}
  
  
  template <typename T>
  Levenberg_Marquardt<T>::Levenberg_Marquardt(){
    cudimotOptions& opts = cudimotOptions::getInstance();
    max_iterations=opts.iterLevMar.value();
    Marquardt=opts.useMarquardt.value();
  }
  
  template <typename T>
  void Levenberg_Marquardt<T>::run(
				   int nvox, int nmeas,
				   int CFP_size,
				   T* meas,
				   T* params,
				   T* CFP) 
  {
    
    long int amount_shared_mem = 0;
    amount_shared_mem += 4*VOXELS_BLOCK*sizeof(double); // Levenberg parameters
    amount_shared_mem += (nmeas*CFP_size)*sizeof(T); // CFP
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // Parameters
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // Gradient
    amount_shared_mem += (NPARAMS*NPARAMS*VOXELS_BLOCK)*sizeof(T); // Hessian
    amount_shared_mem += 2*VOXELS_BLOCK*sizeof(int); // Levenberg parameters
    amount_shared_mem += (NPARAMS*VOXELS_BLOCK)*sizeof(T); // step
    
    cout << "Shared Memory used in Levenberg-Marquardt kernel: " << amount_shared_mem << endl;
    
    int threads_block = VOXELS_BLOCK * THREADS_VOXEL;
    int nblocks=(nvox/VOXELS_BLOCK);
    if(nvox%VOXELS_BLOCK) nblocks++;
    
    if(Marquardt){
      levenberg_kernel<T,true><<<nblocks,threads_block,amount_shared_mem>>>(nvox,nmeas,CFP_size,meas,params,CFP,max_iterations);
    }else{
      levenberg_kernel<T,false><<<nblocks,threads_block,amount_shared_mem>>>(nvox,nmeas,CFP_size,meas,params,CFP,max_iterations);
    }
    sync_check("Levenberg_Marquardt Kernel");
  }
  
  // Explicit Instantiations of the template
  template class Levenberg_Marquardt<float>;
  template class Levenberg_Marquardt<double>;
}
